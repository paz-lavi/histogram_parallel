#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hw8.h"
#include <stdio.h>

__global__ void addCalculateKernel(const int *image, int *dest,
		unsigned int part_size, int arr_size , int mod) {
	int tid = threadIdx.x;

	int image_start = part_size * tid;
	int image_end = image_start + part_size + ((tid == THREADS -1)? mod :0);
//	if (image_end > arr_size)
//		image_end = arr_size;
	int dest_start = arr_size * tid;
	int dest_end = dest_start + arr_size;

	// zero srtarting counters
	for (int j = dest_start; j < dest_end; j++) {
		dest[j] = 0;
	}

	for (int j = image_start; j < image_end; j++)
		dest[dest_start + image[j]]++;

}

__global__ void addMergeKernel(int *histogram, int *temp_arrays, int arr_size) {
	int tid = threadIdx.x;
	// zero starting counters
	for (int i = 0; i < RANGE_SIZE; i++) {
		histogram[i] = 0;
	}
	// merge results. each thread summarize one cell in each temp array
	//for (int i = 0; i < RANGE_SIZE; i++)
	for (int i = 0; i < THREADS; i++)
		histogram[tid] += temp_arrays[arr_size * i + tid];
}

int* calculateHistogramm(int *image, unsigned int size, int arr_size) {
	int *dev_image = 0;
	int *dev_dest_hist = 0;
	int *dev_histogram = 0;
	int *histogram;
	myIntArrCalloc(&histogram, RANGE_SIZE);
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output).
	cudaStatus = hipMalloc((void**) &dev_image, size * sizeof(int));
	cudaStatus = hipMalloc((void**) &dev_histogram, RANGE_SIZE * sizeof(int));
	cudaStatus = hipMalloc((void**) &dev_dest_hist,
			arr_size * sizeof(int) * (arr_size / 4));

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_image, image, size * sizeof(int),
			hipMemcpyHostToDevice);


	// Launch a kernel on the GPU with one thread for each element.
	addCalculateKernel <<<1, THREADS>>>(dev_image, dev_dest_hist, size/THREADS, arr_size , size%THREADS);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n",
				hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr,
				"hipDeviceSynchronize returned error code %d after launching addCalculateKernel!\n",
				cudaStatus);
		goto Error;
	}

	addMergeKernel <<<1, THREADS>>>(dev_histogram, dev_dest_hist , arr_size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n",
				hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr,
				"hipDeviceSynchronize returned error code %d after launching addCalculateKernel!\n",
				cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(histogram, dev_histogram, RANGE_SIZE * sizeof(int),
			hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	Error: hipFree(dev_image);
	hipFree(dev_histogram);
	hipFree(dev_dest_hist);

	return histogram;
}


